#include "hip/hip_runtime.h"
#include "cuda_helper.h"

__global__ void row_sum_kernel(float *A_out, float *A, uint A_cols, uint A_rows) 
{
    uint x = threadIdx.x + blockIdx.x * blockDim.x; // column index in A_out
    uint y = threadIdx.y + blockIdx.y * blockDim.y; // row index in A_out

    if (x < A_cols && y < A_rows)
    {
        float psum = 0;
        for (int r = 0; r < A_rows; r++)
        {
            uint A_idx = r*A_cols + x;
            psum += A[A_idx];
        }

        A_out[x] = psum;
    }

}

float *row_sum(float *h_A, uint A_rows, uint A_cols)
{
    dim3 dimsA(A_cols, A_rows, 1); 
    dim3 dimsA_out(A_cols, 1, 1);

    uint A_size = dimsA.x * dimsA.y;
    uint A_out_size = dimsA_out.x * dimsA_out.y;

    uint A_mem_size = sizeof(float)*A_size;
    uint A_out_mem_size = sizeof(float)*A_out_size;

    // declare the arrays
    float *h_A_out, *d_A, *d_A_out;

    // allocate host memory
    checkCuda( hipHostMalloc((void**)&h_A_out, A_out_mem_size) );

    // allocate device memory
    checkCuda( hipMalloc((void**)&d_A, A_mem_size) );
    checkCuda( hipMalloc((void**)&d_A_out, A_out_mem_size) );

    // copy host memory to device
    checkCuda( hipMemcpy(d_A, h_A, A_mem_size, hipMemcpyHostToDevice) );

    // define block size and number of blocks (grid size)
    uint block_size = 32;
    dim3 threads(block_size, block_size);

    // calculate number of blocks
    dim3 grid((dimsA_out.x / threads.x)+1, (dimsA_out.y / threads.y)+1);

    // exectute the kernels
    row_sum_kernel<<<grid, threads>>>(d_A_out, d_A, dimsA.x, dimsA.y);

    // copy result from device to host
    checkCuda( hipMemcpy(h_A_out, d_A_out, A_out_mem_size, hipMemcpyDeviceToHost) );    

    checkCuda( hipFree(d_A) );
    checkCuda( hipFree(d_A_out) );

    return h_A_out;
}