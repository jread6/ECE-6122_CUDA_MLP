#include "hip/hip_runtime.h"
#include "cuda_helper.h"

__global__ void clip_gradients_kernel(float *A, uint A_cols, uint A_rows) 
{
    uint x = threadIdx.x + blockIdx.x * blockDim.x; // column index in A
    uint y = threadIdx.y + blockIdx.y * blockDim.y; // row index in A

    if (x < A_cols && y < A_rows)
    {
        uint A_idx = y*A_cols + x;
        if (A[A_idx] < -1)
        {
            A[A_idx] = -1;
        }
        else if (A[A_idx] > 1 || std::isnan(A[A_idx]))
        {
            A[A_idx] = 1;
        }
    }
}

float *clip_gradients(float* h_A, uint A_rows, uint A_cols)
{
    // B is a [1 x A_cols] vector
    dim3 dimsA(A_cols, A_rows, 1); 

    uint A_size = dimsA.x * dimsA.y;

    uint A_mem_size = sizeof(float)*A_size;

    // declare the arrays
    float *d_A;
    
    // allocate host memory
    // checkCuda( hipHostMalloc((void**)&h_A_out, A_out_mem_size) );

    // allocate device memory
    checkCuda( hipMalloc((void**)&d_A, A_mem_size) );

    // copy host memory to device
    checkCuda( hipMemcpy(d_A, h_A, A_mem_size, hipMemcpyHostToDevice) );

    // define block size and number of blocks (grid size)
    uint block_size = 32;
    dim3 threads(block_size, block_size);

    // calculate number of blocks
    dim3 grid((dimsA.x / threads.x)+1, (dimsA.y / threads.y)+1);

    // exectute the kernels
    clip_gradients_kernel<<<grid, threads>>>(d_A, dimsA.x, dimsA.y);

    // copy result from device to host
    checkCuda( hipMemcpy(h_A, d_A, A_mem_size, hipMemcpyDeviceToHost) );    

    checkCuda( hipFree(d_A) );

    return h_A;
}