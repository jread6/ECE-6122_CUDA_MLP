#include "hip/hip_runtime.h"
#include "cuda_helper.h"

__global__ void dLoss_kernel(float *A, uint *labels, uint A_cols, uint A_rows) 
{
    uint x = threadIdx.x + blockIdx.x * blockDim.x; // column index in A
    uint y = threadIdx.y + blockIdx.y * blockDim.y; // row index in A

    if (x < A_cols && y < A_rows)
    {
        uint label_idx = labels[y]; // labels vector give an integer 0-9
        uint A_idx = y*A_cols + x;

        if (label_idx == x)
        {
            if (A[A_idx] == 0) { A[A_idx] = 1000; }
            else { A[A_idx] = -1/A[A_idx]; }
        }
        else
        {
            A[A_idx] = 0;
        }
    }
}

float *dLoss(float *h_A, uint *labels, uint A_rows, uint A_cols)
{
    dim3 dimsA(A_cols, A_rows, 1); 
    dim3 dimsA_out(A_cols, A_rows, 1);
    dim3 dimsL(1, A_rows, 1);

    uint A_size = dimsA.x * dimsA.y;
    uint A_out_size = dimsA_out.x * dimsA_out.y;
    uint L_size = dimsL.x * dimsL.y;

    uint A_mem_size = sizeof(float)*A_size;
    uint A_out_mem_size = sizeof(float)*A_out_size;
    uint L_mem_size = sizeof(uint)*L_size;

    // declare the arrays
    float *h_A_out, *d_A;
    uint *d_L;

    // allocate host memory
    checkCuda( hipHostMalloc((void**)&h_A_out, A_out_mem_size) );

    // allocate device memory
    checkCuda( hipMalloc((void**)&d_A, A_mem_size) );
    checkCuda( hipMalloc((void**)&d_L, L_mem_size) );

    // copy host memory to device
    checkCuda( hipMemcpy(d_A, h_A, A_mem_size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_L, labels, L_mem_size, hipMemcpyHostToDevice) );

    // define block size and number of blocks (grid size)
    uint block_size = 32;
    dim3 threads(block_size, block_size);

    // calculate number of blocks
    dim3 grid((dimsA.x / threads.x)+1, (dimsA.y / threads.y)+1);

    // exectute the kernels
    dLoss_kernel<<<grid, threads>>>(d_A, d_L, dimsA.x, dimsA.y);

    // copy result from device to host
    checkCuda( hipMemcpy(h_A_out, d_A, A_mem_size, hipMemcpyDeviceToHost) );    

    checkCuda( hipFree(d_A) );
    checkCuda( hipFree(d_L) );

    return h_A_out;
}