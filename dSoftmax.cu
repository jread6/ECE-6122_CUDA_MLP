#include "hip/hip_runtime.h"
#include "cuda_helper.h"

__global__ void dsoftmax_kernel(float *A, uint *labels, uint A_cols, uint A_rows) 
{
    uint x = threadIdx.x + blockIdx.x * blockDim.x; // column index in A
    uint y = threadIdx.y + blockIdx.y * blockDim.y; // row index in A


    if (x < A_cols && y < A_rows)
    {
        uint label = labels[y];
        uint A_idx = y*A_cols + x;

        if (label == x)
        {
            A[A_idx] = (A[A_idx] - 1) / A_rows;
        }
        else
        {
            A[A_idx] = A[A_idx] / A_rows;
        }

    }
}

float *dsoftmax(float* h_A, uint *labels, uint A_rows, uint A_cols)
{
    // B is a [1 x A_cols] vector
    dim3 dimsA(A_cols, A_rows, 1); 
    dim3 dimsA_out(A_cols, A_rows, 1); 
    dim3 dimsL(1, A_rows, 1);

    uint A_size = dimsA.x * dimsA.y;
    uint A_out_size = dimsA_out.x * dimsA_out.y;
    uint L_size = dimsL.x * dimsL.y;

    uint A_mem_size = sizeof(float)*A_size;
    uint A_out_mem_size = sizeof(float)*A_out_size;
    uint L_mem_size = sizeof(uint)*L_size;

    // declare the arrays
    float *h_A_out, *d_A; 
    uint *d_L;

    // allocate host memory
    checkCuda( hipHostMalloc((void**)&h_A_out, A_out_mem_size) );

    // allocate device memory
    checkCuda( hipMalloc((void**)&d_A, A_mem_size) );
    checkCuda( hipMalloc((void**)&d_L, L_mem_size) );

    // copy host memory to device
    checkCuda( hipMemcpy(d_A, h_A, A_mem_size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_L, labels, L_mem_size, hipMemcpyHostToDevice) );

    // define block size and number of blocks (grid size)
    uint block_size = 32;
    dim3 threads(block_size, block_size);

    // calculate number of blocks
    dim3 grid((dimsA_out.x / threads.x)+1, (dimsA_out.y / threads.y)+1);

    // exectute the kernels
    dsoftmax_kernel<<<grid, threads>>>(d_A, d_L, dimsA.x, dimsA.y);

    // copy result from device to host
    checkCuda( hipMemcpy(h_A_out, d_A, A_mem_size, hipMemcpyDeviceToHost) );    

    checkCuda( hipFree(d_A) );
    checkCuda( hipFree(d_L) );

    return h_A_out;
}